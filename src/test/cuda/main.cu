#include "helper.cuh"
#include "matmul.cuh"
#include <cstddef>
#include <cstdint>
#include <cstdlib>
#include <hipblas.h>
#include <iostream>
#include <vector>

#define FLOAT_EQUAL(a, b) (a - b < 1e-1 && a - b > -1e-1)
class MutAnalogMatrixResult {
public:
    MutAnalogMatrixResult(int M, int N)
        : M(M)
        , N(N) {
        c.resize(M * N);
        CUDA_CHECK(hipMalloc(&d_C, sizeof(float) * M * N));
        CUDA_CHECK(hipMemset(d_C, 0, sizeof(float) * M * N));
    }

    bool operator==(const MutAnalogMatrixResult &other) const {
        if (M != other.M || N != other.N) {
            return false;
        }
        // bool res = true;
        for (auto i = 0; i < M * N; i++) {
            if (!FLOAT_EQUAL(c[ i ], other.c[ i ])) {
                std::cout << "c[" << i << "]:" << c[ i ] << " other.c[" << i << "]:" << other.c[ i ]
                          << std::endl;
                // res = res ? false : res;
                return false;
            }
        }
        return true;
    }

    void SaveResult() {
        CUDA_CHECK(hipMemcpy(c.data(), d_C, sizeof(float) * M * N, hipMemcpyDeviceToHost));
    }

    ~MutAnalogMatrixResult() {
        CUDA_CHECK(hipFree(d_C));
    }

public:
    float *d_C;

private:
    std::vector<float> c;
    int M;
    int N;
};

class MutAnalogMatrix {
public:
    MutAnalogMatrix(int M, int N, int K)
        : M(M)
        , N(N)
        , K(K) {
        a.reserve(M * K);
        b.reserve(K * N);
        for (auto i = 0; i < M * K; i++) {
            a.push_back(1.0f + rand() % 10 / 10.0f);
            // a.push_back(1.0f);
        }
        for (auto i = 0; i < N * K; i++) {
            b.push_back(2.0f + rand() % 20 / 20.0f);
            // b.push_back(2.0f);
        }
        CUDA_CHECK(hipMalloc(&d_A, sizeof(float) * M * K));
        CUDA_CHECK(hipMalloc(&d_B, sizeof(float) * K * N));
        CUDA_CHECK(hipMemcpy(d_A, a.data(), sizeof(float) * M * K, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_B, b.data(), sizeof(float) * N * K, hipMemcpyHostToDevice));
    }

    ~MutAnalogMatrix() {
        CUDA_CHECK(hipFree(d_A));
        CUDA_CHECK(hipFree(d_B));
    }

public:
    float *d_A;
    float *d_B;

private:
    std::vector<float> a;
    std::vector<float> b;
    int M;
    int N;
    int K;
};

int main() {
    constexpr int M = 128, N = 128, K = 128;
    GpuTimer timer;
    int repeat_times = 50;
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasHandle_t handle;
    if (hipblasCreate(&handle)) {
        std::cerr << "Create cublas handle error." << std::endl;
        exit(EXIT_FAILURE);
    };
#if 1
#if 0
    std::cout << "sgemm_share \n";
    for (auto i = 0; i < 6; i++) {
        uint64_t m = M << i;
        uint64_t n = N << i;
        uint64_t k = K << i;
        int64_t flops = 2 * m * n * k;
        MutAnalogMatrix mat(m, n, k);
        MutAnalogMatrixResult rmat(m, n);
        timer.start();
        dim3 blockDim(32, 32);
        dim3 gridDim((m + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);
        int tmp_repeat_times = repeat_times;
        for (; tmp_repeat_times--;) {
            sgemm_share<<<gridDim, blockDim>>>(m, n, k, alpha, mat.d_A, mat.d_B, beta,
                rmat.d_C);
        }
        hipDeviceSynchronize();
        timer.stop();
        auto &&elapsed_time = timer.elapsed_millis();
        std::cout << "mut size :" << m << " " << "Time: " << elapsed_time / repeat_times
                  << "ms GFLOPS:" << (flops * repeat_times * 1e-9) / (elapsed_time / 1000)
                  << std::endl;
    }

    std::cout << "sgemm_gmem_coalesce \n";
    for (auto i = 0; i < 6; i++) {
        uint64_t m = M << i;
        uint64_t n = N << i;
        uint64_t k = K << i;
        int64_t flops = 2 * m * n * k;
        MutAnalogMatrix mat(m, n, k);
        MutAnalogMatrixResult rmat(m, n);
        dim3 blockDim(32 * 32);
        dim3 gridDim((m + 32 - 1) / 32, (n + 32 - 1) / 32);
        int tmp_repeat_times = repeat_times;
        timer.start();

        for (; tmp_repeat_times--;) {
            sgemm_gmem_coalesce<32>
                <<<gridDim, blockDim>>>(m, n, k, 1.0f, mat.d_A, mat.d_B, 0.0f, rmat.d_C);
        }
        hipDeviceSynchronize();
        timer.stop();
        auto &&elapsed_time = timer.elapsed_millis();
        std::cout << "mut size :" << m << " " << "Time: " << elapsed_time / repeat_times
                  << "ms GFLOPS:" << (flops * repeat_times * 1e-9) / (elapsed_time / 1000)
                  << std::endl;
    }

    std::cout << "sgemm_1d_blocktiling \n";
    for (auto i = 0; i < 6; i++) {
        uint64_t m = M << i;
        uint64_t n = N << i;
        uint64_t k = K << i;
        int64_t flops = 2 * m * n * k;
        const uint TM = 8;
        const uint BM = 32;
        const uint BN = 32;
        const uint BK = 32;
        MutAnalogMatrix mat(m, n, k);
        MutAnalogMatrixResult rmat(m, n);
        dim3 blockDim(BM * BN / TM);
        dim3 gridDim((n + BN - 1) / BN, (m + BM - 1) / BM);
        int tmp_repeat_times = repeat_times;
        timer.start();

        for (; tmp_repeat_times--;) {
            sgemm_1d_blocktiling<TM, BM, BN, BK>
                <<<gridDim, blockDim>>>(m, n, k, 1.0f, mat.d_A, mat.d_B, 0.0f, rmat.d_C);
        }
        hipDeviceSynchronize();
        timer.stop();
        auto &&elapsed_time = timer.elapsed_millis();
        std::cout << "mut size :" << m << " " << "Time: " << elapsed_time / repeat_times
                  << "ms GFLOPS:" << (flops * repeat_times * 1e-9) / (elapsed_time / 1000)
                  << std::endl;
    }

#endif

    std::cout << "sgemm_2d_blocktiling \n";
    for (auto i = 0; i < 6; i++) {
        uint64_t m = M << i;
        uint64_t n = N << i;
        uint64_t k = K << i;
        int64_t flops = 2 * m * n * k;
        const uint TM = 8;
        const uint TN = 8;
        const uint BM = 128;
        const uint BN = 128;
        const uint BK = 8;
        MutAnalogMatrix mat(m, n, k);
        MutAnalogMatrixResult rmat(m, n);
        dim3 blockDim(BM * BN / TM / TN);
        dim3 gridDim((n + BN - 1) / BN, (m + BM - 1) / BM);
        int tmp_repeat_times = repeat_times;
        timer.start();

        for (; tmp_repeat_times--;) {
            sgemm_2d_blocktiling<TM, TN, BM, BN, BK>
                <<<gridDim, blockDim>>>(m, n, k, alpha, mat.d_A, mat.d_B, beta, rmat.d_C);
        }
        hipDeviceSynchronize();
        timer.stop();
        auto &&elapsed_time = timer.elapsed_millis();
        std::cout << "mut size :" << m << " " << "Time: " << elapsed_time / repeat_times
                  << "ms GFLOPS:" << (flops * repeat_times * 1e-9) / (elapsed_time / 1000)
                  << std::endl;
    }

    std::cout << "demo sgemm_2d_blocktiling_with_transpose \n";
    for (auto i = 0; i < 6; i++) {
        uint64_t m = M << i;
        uint64_t n = N << i;
        uint64_t k = K << i;
        int64_t flops = 2 * m * n * k;
        const uint TM = 8;
        const uint TN = 8;
        const uint BM = 128;
        const uint BN = 128;
        const uint BK = 8;
        MutAnalogMatrix mat(m, n, k);
        MutAnalogMatrixResult rmat(m, n);
        dim3 blockDim(BM * BN / TM / TN);
        dim3 gridDim((n + BN - 1) / BN, (m + BM - 1) / BM);
        int tmp_repeat_times = repeat_times;
        timer.start();

        for (; tmp_repeat_times--;) {
            sgemm_2d_blocktiling_with_transpose<TM, TN, BM, BN, BK>
                <<<gridDim, blockDim>>>(m, n, k, alpha, mat.d_A, mat.d_B, beta, rmat.d_C);
        }
        hipDeviceSynchronize();
        timer.stop();
        auto &&elapsed_time = timer.elapsed_millis();
        std::cout << "mut size :" << m << " " << "Time: " << elapsed_time / repeat_times
                  << "ms GFLOPS:" << (flops * repeat_times * 1e-9) / (elapsed_time / 1000)
                  << std::endl;
    }

    std::cout << "demo sgemm_2d_blocktiling \n";
    for (auto i = 0; i < 6; i++) {
        uint64_t m = M << i;
        uint64_t n = N << i;
        uint64_t k = K << i;
        int64_t flops = 2 * m * n * k;
        const uint TM = 8;
        const uint TN = 8;
        const uint BM = 128;
        const uint BN = 128;
        const uint BK = 8;
        MutAnalogMatrix mat(m, n, k);
        MutAnalogMatrixResult rmat(m, n);
        dim3 blockDim(BM * BN / TM / TN);
        dim3 gridDim((n + BN - 1) / BN, (m + BM - 1) / BM);
        int tmp_repeat_times = repeat_times;
        timer.start();

        for (; tmp_repeat_times--;) {
            sgemm2DBlocktiling<BM, BN, BK, TM, TN>
                <<<gridDim, blockDim>>>(m, n, k, alpha, mat.d_A, mat.d_B, beta, rmat.d_C);
        }
        hipDeviceSynchronize();
        timer.stop();
        auto &&elapsed_time = timer.elapsed_millis();
        std::cout << "mut size :" << m << " " << "Time: " << elapsed_time / repeat_times
                  << "ms GFLOPS:" << (flops * repeat_times * 1e-9) / (elapsed_time / 1000)
                  << std::endl;
    }

    std::cout << "hipblasGemmEx \n";
    for (auto i = 0; i < 6; i++) {
        uint64_t m = M << i;
        uint64_t n = N << i;
        uint64_t k = K << i;
        int64_t flops = 2 * m * n * k;
        MutAnalogMatrix mat(m, n, k);
        MutAnalogMatrixResult rmat(m, n);
        timer.start();
        int tmp_repeat_times = repeat_times;
        for (; tmp_repeat_times--;) {
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, mat.d_B, HIP_R_32F, n,
                         mat.d_A, HIP_R_32F, k, &beta, rmat.d_C, HIP_R_32F, n, HIPBLAS_COMPUTE_32F,
                         CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        }
        hipDeviceSynchronize();
        timer.stop();
        auto &&elapsed_time = timer.elapsed_millis();
        std::cout << "mut size :" << m << " " << "Time: " << elapsed_time / repeat_times
                  << "ms GFLOPS:" << (flops * repeat_times * 1e-9) / (elapsed_time / 1000)
                  << std::endl;
    }

#endif

    //// 正确与否判断
    {
        ///// 1024 矩阵
        for (auto i = 0; i < 6; i++) {
            uint64_t m = M << i;
            uint64_t n = N << i;
            uint64_t k = K << i;

            MutAnalogMatrix mat(m, n, k);
            MutAnalogMatrixResult mata(m, n);

            {
                hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, mat.d_B, HIP_R_32F,
                             n, mat.d_A, HIP_R_32F, k, &beta, mata.d_C, HIP_R_32F, n,
                             HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
                hipDeviceSynchronize();
                mata.SaveResult();
            }
#if 0
            {
                MutAnalogMatrixResult matb(m, n);
                const uint TM = 8;
                const uint BM = 64;
                const uint BN = 64;
                const uint BK = 8;
                dim3 blockDim(BM * BN / TM);
                dim3 gridDim((n + BN - 1) / BN, (m + BM - 1) / BM);
                sgemm_1d_blocktiling<TM, BM, BN, BK>
                    <<<gridDim, blockDim>>>(m, n, k, alpha, mat.d_A, mat.d_B, beta, matb.d_C);
                hipDeviceSynchronize();
                matb.SaveResult();
                if (mata == matb)
                    std::cout << m << " : hipblasGemmEx and sgemm_1d_blocktiling is equal"
                              << std::endl;
                else
                    std::cout << m << " : hipblasGemmEx and sgemm_1d_blocktiling is not equal"
                              << std::endl;
            }
#endif

            {
                MutAnalogMatrixResult matb(m, n);
                const uint TM = 8;
                const uint TN = 8;
                const uint BM = 128;
                const uint BN = 128;
                const uint BK = 32;
                dim3 blockDim(BM * BN / TM / TN);
                dim3 gridDim((n + BN - 1) / BN, (m + BM - 1) / BM);
                sgemm_2d_blocktiling<TM, TN, BM, BN, BK>
                    <<<gridDim, blockDim>>>(m, n, k, alpha, mat.d_A, mat.d_B, beta, matb.d_C);
                CUDA_CHECK_LAST_ERROR();
                hipDeviceSynchronize();
                matb.SaveResult();
                if (mata == matb)
                    std::cout << m << " : hipblasGemmEx and sgemm_2d_blocktiling is equal"
                              << std::endl;
                else
                    std::cout << m << " : hipblasGemmEx and sgemm_2d_blocktiling is not equal"
                              << std::endl;
            }

            {
                MutAnalogMatrixResult matb(m, n);
                const uint TM = 8;
                const uint TN = 8;
                const uint BM = 128;
                const uint BN = 128;
                const uint BK = 32;
                dim3 blockDim(BM * BN / TM / TN);
                dim3 gridDim((n + BN - 1) / BN, (m + BM - 1) / BM);
                sgemm_2d_blocktiling_with_transpose<TM, TN, BM, BN, BK>
                    <<<gridDim, blockDim>>>(m, n, k, alpha, mat.d_A, mat.d_B, beta, matb.d_C);
                CUDA_CHECK_LAST_ERROR();
                hipDeviceSynchronize();
                matb.SaveResult();
                if (mata == matb)
                    std::cout << m
                              << " : hipblasGemmEx and sgemm_2d_blocktiling_with_transpose is equal"
                              << std::endl;
                else
                    std::cout
                        << m
                        << " : hipblasGemmEx and sgemm_2d_blocktiling_with_transpose is not equal"
                        << std::endl;
            }
#if 0
            {
                MutAnalogMatrixResult matc(m, n);
                dim3 blockDim(32 * 32);
                dim3 gridDim((n + 32 - 1) / 32, (m + 32 - 1) / 32);
                sgemm_share<<<gridDim, blockDim>>>(m, n, k, alpha, mat.d_A, mat.d_B, beta,
                                                   matc.d_C);
                hipDeviceSynchronize();
                matc.SaveResult();
                if (mata == matc)
                    std::cout << m << " : hipblasGemmEx and sgemm_share is equal" << std::endl;
                else
                    std::cout << m << " : hipblasGemmEx and sgemm_share is not equal" << std::endl;
            }
            {

                MutAnalogMatrixResult matd(m, n);
                dim3 blockDim(32 * 32);
                dim3 gridDim((m + 32 - 1) / 32, (n + 32 - 1) / 32);
                sgemm_gmem_coalesce<32>
                    <<<gridDim, blockDim>>>(m, n, k, alpha, mat.d_A, mat.d_B, beta, matd.d_C);
                hipDeviceSynchronize();
                matd.SaveResult();
                if (mata == matd)
                    std::cout << m << " : hipblasGemmEx and sgemm_gmem_coalesce is equal"
                              << std::endl;
                else
                    std::cout << m << " : hipblasGemmEx and sgemm_gmem_coalesce is not equal"
                              << std::endl;
            }

#endif
        }
    }

    return 0;
}